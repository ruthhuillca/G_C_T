#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "CImg.h"

#define a_row 299
#define a_col 10
#define ab_colrow 325
#define b_col 10
#define f_dim 3
#define tam f_dim/2
#define BLOCK_SIZE 16

using namespace std;
using namespace cimg_library;


void fill_Matrix(double **&matrix, int m, int n){
  for (size_t i = 0; i < m; i++)
    for (size_t j = 0; j < n; j++) {
      matrix[i][j] = 2;//rand()%2+1;
  }
}

void fill_filter5(double **&matrix){
  matrix[0][0]=1.0 ; matrix[0][1]=4.0 ; matrix[0][2]=6.0; matrix[0][3]=4.0; matrix[0][4]=1.0;
  matrix[1][0]=4.0 ; matrix[1][1]=16.0 ; matrix[1][2]=24.0; matrix[1][3]=16.0; matrix[1][4]=4.0;
  matrix[2][0]=6.0 ; matrix[2][1]=24.0 ; matrix[2][2]=-476.0; matrix[2][3]=24.0; matrix[2][4]=6.0;
  matrix[3][0]=4.0 ; matrix[3][1]=16.0 ; matrix[3][2]=24.0; matrix[3][3]=16.0; matrix[3][4]=4.0;
  matrix[4][0]=1.0 ; matrix[4][1]=4.0 ; matrix[4][2]=6.0; matrix[4][3]=4.0; matrix[4][4]=1.0;
}

void fill_filter3(double **&matrix){
  matrix[0][0]=0.0 ; matrix[0][1]=-1.0 ; matrix[0][2]=0.0;
  matrix[1][0]=-1.0 ; matrix[1][1]=5 ; matrix[1][2]=-1.0;
  matrix[2][0]=0.0 ; matrix[2][1]=-1.0 ; matrix[2][2]=0.0;
}
  /*
  void fill_filter3(double **&matrix){
    matrix[0][0]=1 ; matrix[0][1]=1 ; matrix[0][2]=1;
    matrix[1][0]=1 ; matrix[1][1]=1 ; matrix[1][2]=1;
    matrix[2][0]=1 ; matrix[2][1]=1 ; matrix[2][2]=1;
  }*/


void print_Matrix(double **&matrix, int m, int n){
  for (size_t i = 0; i < m; i++){
    for (size_t j = 0; j < n; j++)
      printf("%.2f\t", matrix[i][j] );
    printf("\n");
  }
}

void createSimpleMatrix(double **&matrix, int row, int col){
  matrix = (double**)malloc(row*sizeof(double));
  for(size_t i=0; i<row; ++i)
    matrix[i] = (double*)malloc(col*sizeof(double));
}

void createMatrixs(double **&matrix_h, double **&matrix_d, double **&matrix_dd, size_t totalsize, int m, int n){
  size_t row_size = sizeof(double*)*m;
  // PARA EL HOST
  matrix_h = (double**)malloc(row_size);
  matrix_h[0] = (double*)malloc(totalsize);
  // PARA EL DEVICE
  hipMalloc((void**)&matrix_d, row_size);
  matrix_dd = (double**)malloc(row_size);
  hipMalloc((void**)&matrix_dd[0],totalsize);

  for (size_t i = 1; i < m; i++) {
    matrix_h[i] = matrix_h[i-1] + n;
    matrix_dd[i] = matrix_dd[i-1] + n;
  }
  hipMemcpy(matrix_d, matrix_dd, row_size, hipMemcpyHostToDevice);
}

void convolucion(double **matrix, int m_row, int m_col, double **filter, double fil_const, double fil_dim, double  **&out){

  for (int i = 0; i < m_row; i++) {
    for (int j = 0; j < m_col; j++) {
      double res = 0.0;
      int radio_i = -fil_dim/2;
      for (int i_f = 0; (i_f<fil_dim) && (radio_i<=(fil_dim/2)); ++i_f, ++radio_i) {
        int radio_j = -fil_dim/2;
        for (int j_f = 0; (j_f<fil_dim) &&(radio_j<=(fil_dim/2)); ++j_f, ++radio_j) {
          if( m_row >(i+radio_i) && (i+radio_i)>=0 && m_col>(j+radio_j) && (j+radio_j)>=0){
            res += matrix[i+radio_i][j+radio_j] * filter[i_f][j_f];
          }
        }
      }
      out[i][j] = res/fil_const;
    }
  }
}

__global__ void  convolucion_cuda(double **matrix, int m_row, int m_col,double **filter, double fil_const, int fil_dim , double **out){
  int row = threadIdx.x + blockIdx.x*blockDim.x;
  int col = threadIdx.y + blockIdx.y*blockDim.y;
  if( row<m_row && col < m_col){
    double res = 0.0;
    int radio_i = -fil_dim/2;
    for (int i = 0; i < fil_dim; ++i){
      if(radio_i <= fil_dim/2){
        int radio_j = -fil_dim/2;
        for (int j = 0; j < fil_dim ; ++j){
          if(radio_j<=fil_dim/2){
            if( m_row>(row+radio_i) && (row+radio_i)>=0 && m_col>(col+radio_j) && (col+radio_j)>=0){
              res += matrix[row+radio_i][col+radio_j] * filter[i][j];
            }
          }
          radio_j++;
        }
      }
      radio_i++;
    }
    out[row][col] = res/fil_const ;
  }
}

void colorTOgrayMatrix( CImg<unsigned char> &color, double **&gray){
    for(int i=0;i<color.height(); ++i){
        for(int j=0; j<color.width(); ++j){
            int R = (int)color(i,j,0,0);
            int G = (int)color(i,j,0,1);
            int B = (int)color(i,j,0,2);
            int grayValue = (int)(0.33*R + 0.33*G + 0.33*B);
            //tmp(x, y, 0,0 ) = grayValue;
            gray[i][j] = grayValue;
        }
    }
}

void print_RGB(CImg<unsigned char> color){
    for(int i=0;i<color.height(); ++i){
        for(int j=0; j<color.width(); ++j){
            printf("%d\t%d\t\t%d\t%d\t%d\n",i,j,(int)color(i,j,0,0),(int)color(i,j,0,1), (int)color(i,j,0,2));
        }
    }
}

CImg<unsigned char> matrixTOgray( double **matrix, int row, int col){
    CImg<unsigned char> tmp(col,row);
    for(int i=0; i<row; ++i)
        for(int j=0;j<col; ++j){
            tmp(i,j,0,0) = (int)matrix[i][j];
        }
    return tmp;
}

void grayConvolucion(CImg<unsigned char> &imagen, double fil_const, int fil_dim){
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((imagen.height()-1)/BLOCK_SIZE+1,((imagen.width()-1)/BLOCK_SIZE)+1);
    
    size_t total_size = sizeof(double)*imagen.width()*imagen.height();
    size_t filter_size = sizeof(double)*fil_dim*fil_dim;
    int row = imagen.width(), col = imagen.height();
    
    double **in, **de_1, **tmp_1;
    double **out, **de_2, **tmp_2;
    double **filter, **de_f, **tmp_f;
    
    createMatrixs(in, de_1, tmp_1, total_size, row, col);
    createMatrixs(out, de_2, tmp_2, total_size, row, col);
    createMatrixs(filter, de_f, tmp_f, filter_size, fil_dim, fil_dim);
    
    colorTOgrayMatrix(imagen, in);
    
    if(fil_dim == 3){
        fill_filter3(filter);
    }else if(fil_dim == 5)
        fill_filter5(filter);
    else{
        printf("Error en la dimension del filtro\n");
        return;
    }
    hipMemcpy(tmp_1[0], in[0], total_size, hipMemcpyHostToDevice);
    hipMemcpy(tmp_f[0], filter[0], filter_size, hipMemcpyHostToDevice);
    
    convolucion_cuda<<<blocksPerGrid,threadsPerBlock>>>(de_1, row, col, de_f, fil_const, fil_dim, de_2);
    hipDeviceSynchronize();
    
    hipMemcpy(out[0], tmp_2[0], total_size, hipMemcpyDeviceToHost);
    
    CImg<unsigned char> salida;
    salida = matrixTOgray(out, row, col);
    salida.display("salida");
}


void grayConvolucionCPU(CImg<unsigned char> &imagen, double fil_const, int fil_dim){

    double **mat , **filter, **out;
    int row = imagen.width(), col = imagen.height();
    createSimpleMatrix(mat, row, col);
    createSimpleMatrix(filter, fil_dim, fil_dim);
    createSimpleMatrix(out, row, col);

    colorTOgrayMatrix(imagen, mat);

    fill_filter3(filter);

    clock_t t0, t1;
    t0 = clock();
    convolucion(mat,row,col,filter,fil_const,fil_dim,out);
    t1 = clock();
    printf("Tiempo  %.2fs\n",((double)(t1-t0)/CLOCKS_PER_SEC));
    CImg<unsigned char> salida;
    salida = matrixTOgray(out, row, col);
    salida.display("Bye!");
}



int main(int argc, char const *argv[]) {

    CImg<unsigned char> image("ironman.bmp");
    grayConvolucionCPU(image,1 , 3);
    grayConvolucion(image,1 , 3);
    return 0;
}

// nvcc convolu.cu -lX11 -w
// ./a.out